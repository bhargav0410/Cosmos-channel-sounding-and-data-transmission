#include "hip/hip_runtime.h"
#include "RakeReceiver.cuh"

RakeReceiver::RakeReceiver {}

RakeReceiver::~RakeReceiver {}

__global__ void findPeaks(hipFloatComplex* H, float thres1, int* numPeaks, hipFloatComplex* gains, int* delays, int numAnts, int length) {
	float thres = thres1;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	__shared__ int temp = 0;
	
	if (tid < length) {
		if (hipCabsf(H[tid]) < thres) {
			gains[temp] = H[tid];
			delays[temp] = threadIdx.x;
			atomicAdd(&temp,1);
			
		}
	}
	numPeaks[blockIdx.x] = temp;
}

__global__ void combineChanGains(float Hsqrd, int* numPeaks, hipFloatComplex* gains, int numAnts) {
	
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	extern __shared__ temp[];
	int tempID = threadIdx.x;
	
	for (int i = 1; i < numPeaks*numAnts; i = i*2) {
		if (threadIdx.x%(2*i) == 0) {
			temp[tempID] = hipCaddf(temp[tempID],temp[tempID+i]);
		}
	}
	
}

__global__ void multiplyWithChannelConj(hipFloatComplex* Yf, hipFloatComplex* Y, hipFloatComplex* H, int* numPeaks, int numAnts, int length) {
	
	int tid = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x + blockIdx.z*gridDim.y*gridDim.x*blockDim.x; 
	
	if (blockIdx.x*blockDim.x <= (length - blockDim.x)) {
		if () {
			Yf[tid] = 
		}
	}
	
}